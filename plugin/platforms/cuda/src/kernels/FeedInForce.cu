#include "hip/hip_runtime.h"

extern "C" __global__ void computeForceKernel(
unsigned long long* __restrict__ forceBuffer,
const double* __restrict__ forces_in,
real* __restrict__ energyBuffer
) {
    int threadIndex = threadIdx.x;
    for (int index=blockIdx.x * blockDim.x + threadIndex; index<NUM_ATOMS; index+=blockDim.x * gridDim.x) {
        atomicAdd(&forceBuffer[index], static_cast<unsigned long long>((long long)(forces_in[index*3]*0x100000000)));                         // X component of the force
        atomicAdd(&forceBuffer[index+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long)(forces_in[index*3+1]*0x100000000)));      // Y component of the force
        atomicAdd(&forceBuffer[index+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long)(forces_in[index*3+2]*0x100000000)));    // Z component of the force
    }
    if (threadIndex == 0) {
        energyBuffer[0] += 0;
    }
}
